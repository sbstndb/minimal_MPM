#include "hip/hip_runtime.h"
#include "grid.h"
#include <iostream>


__global__ void initializeGridKernel(float* d_x, float* d_y, float *d_z, int nx, int ny, int nz, float h){
	int i = blockIdx.x * blockDim.x + threadIdx.x ; 
        int j = blockIdx.y * blockDim.y + threadIdx.y ;
        int k = blockIdx.z * blockDim.z + threadIdx.z ;
	
	if (i < nx && j < ny && k < nz){
		int index = k * (nx*ny) + j*nx + i ;
		d_x[index] = static_cast<float>(i*h) ; 
		d_y[index] = static_cast<float>(j*h) ;
		d_z[index] = static_cast<float>(k*h);
	}

}

void Grid::initializeGrid(){
	// definition des blocs
	dim3 threadsPerBlocks(8,8,8) ; 
	dim3 blockPerGrid(
		(nx + threadsPerBlocks.x - 1)  / threadsPerBlocks.x ,
                (ny + threadsPerBlocks.y - 1)  / threadsPerBlocks.y ,
                (nz + threadsPerBlocks.z - 1)  / threadsPerBlocks.z 
		
	);
        initializeGridKernel<<<threadsPerBlocks, blockPerGrid>>>(d_x, d_y, d_z, nx, ny, nz, h) ;
        hipDeviceSynchronize();
	std::cout << "grid initialized" << std::endl ; 
};

