#include "hip/hip_runtime.h"
#include <iostream>
#include "math.h"


// Kernel CUDA simple
__global__ void algoKernel() {
    printf("Exécution d'un algorithme sur le GPU.\n");
}

// Fonction hôte
void algoFunction() {
    algoKernel<<<1, 1>>>();
    hipDeviceSynchronize();
}

// Fonction appelée depuis main.cpp
void cudaFunction() {
    mathFunction();
    algoFunction();
}

