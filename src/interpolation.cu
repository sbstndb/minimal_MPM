#include "hip/hip_runtime.h"
#include "interpolation.h"
#include <iostream>
/**
        // define shape functions
        float Interp(float, float, float, float, float, float) ; // switch case in Type
        //
        float LinearInterp(float, float, float, float, float, float) ;
        float GimpInterp(float, float, float, float, float, float) ;
        float CpdiInterp(float, float, float, float, float, float) ;
        float QuadSplineInterp(float, float, float, float, float, float) ;
        float CubicSplineInterp(float, float, float, float, float, float) ;
**/



__device__ float linearInterp(float xp, float yp, float zp, float xn, float yn, float zn, float h){
	// without if condition, its a low level function
	float xd = abs(xp - xn) ; 
	float yd = abs(yp - yn) ;
	float zd = abs(zp - zn) ; 

	float w = xd * yd * zd / h ; 
	return w ; 
};



__global__ void weightInterpKernel(
		float *d_xp, float *d_yp, float *d_zp,
		float *d_xn, float *d_yn, float *d_zn,
		int *d_node_x, int * d_node_y, int * d_node_z,
		float *d_weights, 
		int nParticles, 
		int nx, int ny, int nz, float h){
	// Be careful : only with linear shape function because of d weights
	int idx = blockIdx.x * blockDim.x + threadIdx.x ; 
	if (idx >= nParticles) return ; 
	// now we want to compute 8 weights per particles, stored in d_weigths
	
	// particle positions
	float xp = d_xp[idx] ; 
	float yp = d_yp[idx] ; 
	float zp = d_zp[idx] ; 

	// grid indexes
	int i0 = floorf(xp/h); 
	int j0 = floorf(yp/h);
	int k0 = floorf(zp/h);

	// future : verifier que les indices sont dans les bonnes limites, 
	// avec min(max(i0, 0), nx-2) par ex
	//

	int index_node;
	int index_weight= 0 ; 
	float xn, yn, zn ; 
	float weight;

	for (int i = i0 ; i < i0 + 2 ; i++){
		for (int j = j0 ; j < j0 + 2 ; j++){
			for (int k = k0 ; k < k0 + 2 ; k++){
			        index_node = k * (nx*ny) + j*nx + i;
			        xn = d_xn[index_node];
			        yn = d_yn[index_node] ;
			        zn = d_zn[index_node] ;
			        weight = linearInterp(xp, yp, zp, xn, yn, zn, h);
				d_weights[index_weight] = weight ; 
				index_weight++;
			}
		}
	}

}

void Interpolation::weightInterp(
                float *d_xp, float *d_yp, float *d_zp,
                float *d_xn, float *d_yn, float *d_zn,
		int *d_node_x, int* d_node_y, int *d_node_z ,
                float *d_weights,
                int nParticles,
                int nx, int ny, int nz, float h){

        int threadsPerBlocks = 256 ;
        int blocksPerGrid = (nParticles + threadsPerBlocks -1)/threadsPerBlocks ;

        weightInterpKernel<<<threadsPerBlocks, blocksPerGrid>>>(
                        d_xp, d_yp, d_zp,
                        d_xn, d_yn, d_zn,
			d_node_x, d_node_y, d_node_z,
                        d_weights,
                        nParticles,
                        nx, ny, nz, h) ;

        hipDeviceSynchronize() ;
	std::cout << "Weight Interpolation" << std::endl ; 
};


__global__ void P2GKernel(float *fp, float *fi, float *weights, int nParticles){
        // Be careful : only with linear shape function because of d weights
        int idx = blockIdx.x * blockDim.x + threadIdx.x ;
        if (idx >= nParticles) return ;

}

__global__ void G2PKernel(float *fp, float *fi, float *weights, int nParticles){
        // Be careful : only with linear shape function because of d weights
        int idx = blockIdx.x * blockDim.x + threadIdx.x ;
        if (idx >= nParticles) return ;

}




void P2G(float *fp, float * fi , float * weights, int nParticles){
	// be careful : 
	// TODO : add type of interpolation , spline etc
        int threadsPerBlocks = 256 ;
        int blocksPerGrid = (nParticles + threadsPerBlocks -1)/threadsPerBlocks ;
	P2GKernel<<<threadsPerBlocks, blocksPerGrid>>>(fp, fi, weights, nParticles);

}

void G2P(float *fp, float *fi, float* weights, int nParticles){
        int threadsPerBlocks = 256 ;
        // TODO : add type of interpolation , spline etc	
        int blocksPerGrid = (nParticles + threadsPerBlocks -1)/threadsPerBlocks ;
	G2PKernel<<<threadsPerBlocks, blocksPerGrid>>>(fp, fi, weights, nParticles) ; 

}




