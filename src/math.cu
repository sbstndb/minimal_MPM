
#include <hip/hip_runtime.h>
#include <iostream>
#include "math.h"

// Kernel CUDA simple
__global__ void mathKernel() {
    printf("Calculs mathématiques sur le GPU.\n");
}

// Fonction hôte
void mathFunction() {
    mathKernel<<<1, 1>>>();
    hipDeviceSynchronize();
}

